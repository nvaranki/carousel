
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <algorithm>

__global__ 
void kAdd(int *c, const int *a, const int *b)
{
    const int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ 
void kSub(int *c, const int *a, const int *b)
{
    const int i = threadIdx.x;
    c[i] = a[i] - b[i];
}

__host__
hipError_t setup(int dn)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(dn);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    return cudaStatus;
}

__host__
hipError_t alloc( int** a, int** d, int** dev_a, int** dev_b, int** dev_c, int** dev_d, size_t size )
{
    hipError_t cudaStatus;

    // Allocate host buffers for two vectors

    cudaStatus = hipHostMalloc( a, size * sizeof(int) , hipHostMallocDefault);// in page-locked memory
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "alloc host A failed %u\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipHostMalloc( d, size * sizeof(int) , hipHostMallocDefault);// in page-locked memory
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "alloc host D failed %u\n", cudaStatus);
        return cudaStatus;
    }

    // Allocate GPU buffers for three vectors

    cudaStatus = hipMalloc((void**)dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc device A failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc device B failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc device C failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)dev_d, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc device D failed!");
        return cudaStatus;
    }

    return cudaStatus;
}

__host__
void releaseOnDevice( int* dev_a, int* dev_b, int* dev_c, int* dev_d )
{
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);
}

__host__
void releaseOnHost( int* hst_a, int* hst_d )
{
    hipHostFree(hst_a);
    hipHostFree(hst_d);
}

__host__
void releaseOnHost( int* hst_a, int* hst_d, int* tst_a, int* tst_b, int* tst_c, int* tst_d )
{
    hipHostFree(hst_a);
    hipHostFree(hst_d);
    delete[] tst_a;
    delete[] tst_b;
    delete[] tst_c;
    delete[] tst_d;
}

__host__
hipError_t upload( const int* hst_b, int* dev_b, size_t size )
{
    hipError_t cudaStatus;

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy( dev_b, hst_b, size * sizeof(int), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy B failed!\n");
        return cudaStatus;
    }
    return cudaStatus;
}

__host__
hipError_t upload( const int* hst_a, int* dev_a, size_t size, hipStream_t stream )
{
    hipError_t cudaStatus;

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpyAsync( dev_a, hst_a, size * sizeof(int), hipMemcpyHostToDevice, stream );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy A failed!\n");
        return cudaStatus;
    }
    return cudaStatus;
}

__host__
hipError_t download( 
    int* hst_a, int* hst_b, int* hst_c, int* hst_d, 
    int* dev_a, int* dev_b, int* dev_c, int* dev_d, size_t size )
{
    hipError_t cudaStatus;

    // Copy vectors from GPU buffer to host memory.
    cudaStatus = hipMemcpy( hst_a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy A failed!");
        return cudaStatus;
    }
    cudaStatus = hipMemcpy( hst_b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy B failed!");
        return cudaStatus;
    }
    cudaStatus = hipMemcpy( hst_c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy C failed!");
        return cudaStatus;
    }
    cudaStatus = hipMemcpy( hst_d, dev_d, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy D failed!");
        return cudaStatus;
    }
    return cudaStatus;
}

__host__
hipError_t download( int* hst_a, int* dev_a, size_t size, hipStream_t stream )
{
    hipError_t cudaStatus;

    // Copy input vectors from GPU buffer to host memory.
    cudaStatus = hipMemcpyAsync( hst_a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost, stream );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy A failed!");
        return cudaStatus;
    }
    return cudaStatus;
}

__host__
void fill( int base, int* a, size_t size)
{
    for (int i = 0; i < size; i++)
        a[i] = base + i;
}

// Helper function for using CUDA to process vectors in parallel.
__host__
hipError_t carousel( 
    int* hst_a, int* hst_d, 
    int* dev_a, int* dev_b, int* dev_c, int* dev_d, 
    const unsigned int size, const unsigned int ts, const unsigned int repeat )
{
    hipError_t cudaStatus;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipStream_t sX, sY, sI, sO;
    hipStreamCreate( &sX );
    hipStreamCreate( &sY );
    hipStreamCreate( &sI );
    hipStreamCreate( &sO );

    hipEventRecord(startEvent, 0);

    const unsigned int tse(std::max(1, std::min((int)ts, 1024)));
    dim3 dg(std::max(1, (int)(size/tse))), db(std::min((int)tse, (int)size));

    // Launch synchronized streams on the GPU with one thread for each element.
    for( unsigned int i = 0; i < repeat; i++ )
    {
        // I: ↓X fill upload₀            ↓X fill upload₁            ↓X fill upload₂           ...
        // X:                ↑I ↓Y kAdd₀                ↑I ↓Y kAdd₁                ↑I ↓Y kAdd₂
        // Y:                           ↓O ↑X kSub₀                ↓O ↑X kSub₁                ↓O ↑X kSub₂
        // O:                                      ↑Y download₀               ↑Y download₁               ↑Y download₂

        // I: get and upload next input
        fprintf(stdout, "push %d\n", i);
        hipStreamSynchronize( sX );
        fprintf(stdout, "push %d (sX)\n", i);
        fill( i, hst_a, size ); //TODO get new input data
        upload( hst_a, dev_a, size, sI );

        // X: dev_c = dev_a + dev_b
        fprintf(stdout, "add  %d\n", i);
        hipStreamSynchronize( sI );
        fprintf(stdout, "add  %d (sI)\n", i);
        hipStreamSynchronize( sY );
        fprintf(stdout, "add  %d (sY)\n", i);
        kAdd <<<dg,db,0,sX>>> (dev_c, dev_a, dev_b); 
        
        // Y: dev_d = dev_c - dev_b
        fprintf(stdout, "sub  %d\n", i);
        hipStreamSynchronize( sO );
        fprintf(stdout, "sub  %d (sO)\n", i);
        hipStreamSynchronize( sX );
        fprintf(stdout, "sub  %d (sX)\n", i);
        kSub <<<dg,db,0,sY>>> (dev_d, dev_c, dev_b);
        
        // O: download last result
        fprintf(stdout, "pull %d\n", i);
        hipStreamSynchronize( sY );
        fprintf(stdout, "pull %d (sY)\n", i);
        download( hst_d, dev_d, size, sO );
        //TODO signal out the hst_d is ready
        hipStreamSynchronize(sO);
        const char* fmt = "Result: %c={%4d,%4d,%4d,%4d,%4d} - pull\n";
        printf(fmt, 'D', hst_d[0], hst_d[1], hst_d[2], hst_d[3], hst_d[4]);
    }
    hipDeviceSynchronize(); // waits until all streams finished

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "kAdd-kSub launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kAdd!\n", cudaStatus);
        goto Error;
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float ms; // elapsed time in milliseconds
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    fprintf(stderr, "Time for carousel execute #%d: %.3f us (%.3f us/cycle,%.1f ns/cell)\n", repeat, ms*1000., ms*1000./repeat, ms*1000000./ size /repeat );
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

Error:
    hipStreamDestroy( sX );
    hipStreamDestroy( sY );
    hipStreamDestroy( sI );
    hipStreamDestroy( sO );

    return cudaStatus;
}

__host__
int main()
{
    const int device = 0;
    const int arraySize = /*5;/*/ 3 * 64 * 32; // 3 blocks to compute on 64*32=2048 cores
    fprintf(stderr, "Array size: %d\n", arraySize);
    int* a; // input
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int* d; // output
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;
    int* dev_d = nullptr;
    hipError_t cudaStatus = hipErrorUnknown;

    // Setup and prepare.
    cudaStatus = setup(device);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "setup device %d failed %u\n", device, cudaStatus);
        return 1;
    }

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEventRecord(startEvent, 0);

    cudaStatus = alloc( &a, &d, &dev_a, &dev_b, &dev_c, &dev_d, arraySize );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "alloc failed %u\n", cudaStatus);
        releaseOnDevice( dev_a, dev_b, dev_c, dev_d );
        releaseOnHost( a, d );
        return 2;
    }
    cudaStatus = upload( b, dev_b, arraySize );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "upload failed %u\n", cudaStatus);
        releaseOnDevice( dev_a, dev_b, dev_c, dev_d );
        releaseOnHost( a, d );
        return 3;
    }

    // Add-then-subtract vectors in parallel.
    cudaStatus = carousel( a, d, dev_a, dev_b, dev_c, dev_d, arraySize, 1024, 100000 );
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "carousel failed %u\n", cudaStatus);
        releaseOnDevice( dev_a, dev_b, dev_c, dev_d );
        releaseOnHost( a, d );
        return 4;
    }

    int* tst_a = new int[arraySize];
    int* tst_b = new int[arraySize];
    int* tst_c = new int[arraySize];
    int* tst_d = new int[arraySize];
    cudaStatus = download( tst_a, tst_b, tst_c, tst_d, dev_a, dev_b, dev_c, dev_d, arraySize );
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "upload failed %u\n", cudaStatus);
        releaseOnDevice( dev_a, dev_b, dev_c, dev_d );
        releaseOnHost( a, d, tst_a, tst_b, tst_c, tst_d );
        return 5;
    }

    //TODO compare

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float ms; // elapsed time in milliseconds
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    fprintf(stderr, "Time for the test execute: %.3f ms\n", ms);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    const char* fmt = "Result: %c={%4d,%4d,%4d,%4d,%4d}\n";
    printf(fmt, 'A', tst_a[0], tst_a[1], tst_a[2], tst_a[3], tst_a[4]);
    printf(fmt, 'B', tst_b[0], tst_b[1], tst_b[2], tst_b[3], tst_b[4]);
    printf(fmt, 'C', tst_c[0], tst_c[1], tst_c[2], tst_c[3], tst_c[4]);
    printf(fmt, 'D', tst_d[0], tst_d[1], tst_d[2], tst_d[3], tst_d[4]);

    releaseOnDevice( dev_a, dev_b, dev_c, dev_d );
    releaseOnHost( a, d, tst_a, tst_b, tst_c, tst_d );

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        releaseOnDevice( dev_a, dev_b, dev_c, dev_d );
        releaseOnHost( a, d, tst_a, tst_b, tst_c, tst_d );
        return 6;
    }

    return 0;
}
